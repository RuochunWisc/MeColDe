// Reference Scan implementation - Author: Ananoymous student of ME759 Fall 2017
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/scan.h>


void initializeSize(FILE* fp, int* nVertices, int* nFaces)
{
	char x[1024];
	while (fscanf(fp, "%1023s", x)) {
        	if (strcmp(x,"ply")==0) continue;
		if ((strcmp(x,"format")==0)||(strcmp(x,"comment")==0)||(strcmp(x,"property")==0)) {fgets(x, 1023, fp);continue;}
								//	{fscanf(fp,"%[^\n]", x);continue;}
		if (strcmp(x,"element")==0) {
			fscanf(fp, "%1023s", x);
			if (strcmp(x,"vertex")==0) fscanf(fp, "%d\n", &nVertices[0]);
			else if (strcmp(x,"face")==0) fscanf(fp, "%d\n", &nFaces[0]);
		}
		if (strcmp(x,"end_header")==0) break;
    	}
}

void initializeArr(FILE* fp, double* ArrVerX, double* ArrVerY, double* ArrVerZ, int* ArrFace, int V, int F)
{
	int i;
	char x[1024];	
	for(i=0; i<V; ++i){
		fscanf(fp,"%f",&ArrVerX[i]);
		fscanf(fp,"%f",&ArrVerY[i]);
		fscanf(fp,"%f",&ArrVerZ[i]);
		fgets(x, 1023, fp);
//		if(r == EOF){
//			rewind(fp);
//		}
	}

	for (i=0; i<F; ++i){
		fscanf(fp, "%1023s", x);
		fscanf(fp, "%d", &ArrFace[3*i]);
		fscanf(fp, "%d", &ArrFace[3*i+1]);
		fscanf(fp, "%d", &ArrFace[3*i+2]);
	}
}

bool intersect(int i, int j, double *p_x, double *p_y, double *p_z, double *tri)
{



}

double get_volume(int num_1, int num_2, double* faces_1, double* faces_2)
{


}

int main(int argc, char* argv[]) {
	FILE *fp_1 = fopen("bun_zipper.ply","r");
	FILE *fp_2 = fopen("test_mesh.ply","r");
	//allocate resources
	int nV_1=0, nF_1=0, nV_2=0, nF_2=0,i,j;
	int num_ray_intersect;
	float time = 0.f;
	initializeSize(fp_1,&nV_1, &nF_1);
	initializeSize(fp_2,&nV_2, &nF_2);
	double *vertices_x_1= (double *)malloc(sizeof(double)*nV_1);
	double *vertices_y_1= (double *)malloc(sizeof(double)*nV_1);
	double *vertices_z_1= (double *)malloc(sizeof(double)*nV_1);
        int *faces_1   = (int *)malloc(sizeof(int)*nF_1*3);
	double *vertices_x_2= (double *)malloc(sizeof(double)*nV_2);
        double *vertices_y_2= (double *)malloc(sizeof(double)*nV_2);
        double *vertices_z_2= (double *)malloc(sizeof(double)*nV_2);
        int *faces_2   = (int *)malloc(sizeof(int)*nF_2*3);

	initializeArr(fp_1,vertices_x_1,vertices_y_1,vertices_z_1,faces_1,nV_1,nF_1);
	initializeArr(fp_2,vertices_x_2,vertices_y_2,vertices_z_2,faces_2,nV_2,nF_2);
	// Your code here
 
  	double *face_coord_1 = (double *)malloc(sizeof(double)*nF_1*9);
	double *face_coord_2 = (double *)malloc(sizeof(double)*nF_2*9);
 // hipMalloc((void**)&dout,size);
	//hipMalloc((void**)&din,size);
	int *inside_point_set_1 = (int *)malloc(sizeof(int)*nV_1);
	int *inside_point_set_2 = (int *)malloc(sizeof(int)*nV_2);
 
  	hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
  	hipEventRecord(startEvent_inc,0); // starting timing for inclusive  

	for (i=0;i<nF_1;++i){
		face_coord_1[9*i] = vertices_x_1[faces_1[3*i]];
		face_coord_1[9*i+1] = vertices_y_1[faces_1[3*i]];
  		face_coord_1[9*i+2] = vertices_z_1[faces_1[3*i]];
		face_coord_1[9*i+3] = vertices_x_1[faces_1[3*i+1]];
                face_coord_1[9*i+4] = vertices_y_1[faces_1[3*i+1]];
                face_coord_1[9*i+5] = vertices_z_1[faces_1[3*i+1]];
		face_coord_1[9*i+6] = vertices_x_1[faces_1[3*i+2]];
                face_coord_1[9*i+7] = vertices_y_1[faces_1[3*i+2]];
                face_coord_1[9*i+8] = vertices_z_1[faces_1[3*i+2]];
	}

	for (i=0;i<nF_2;++i){
                face_coord_2[9*i] = vertices_x_2[faces_2[3*i]];
                face_coord_2[9*i+1] = vertices_y_2[faces_2[3*i]];
                face_coord_2[9*i+2] = vertices_z_2[faces_2[3*i]];
                face_coord_2[9*i+3] = vertices_x_2[faces_2[3*i+1]];
                face_coord_2[9*i+4] = vertices_y_2[faces_2[3*i+1]];
                face_coord_2[9*i+5] = vertices_z_2[faces_2[3*i+1]];
                face_coord_2[9*i+6] = vertices_x_2[faces_2[3*i+2]];
                face_coord_2[9*i+7] = vertices_y_2[faces_2[3*i+2]];
                face_coord_2[9*i+8] = vertices_z_2[faces_2[3*i+2]];
        }
	//AABB construction
	//NO, dont do this; instead, find points if inside the mesh to determine if this point belongs to the intersection part, this is not likely to be wrong in such a convex struct
	//how to determine if a point is inside the mesh? a lite way, find the closet point on the mesh(para), then move the point towards the direction defined by the point the center of the mesh(the dist to move is around 1/10 of the element length), if the closet point becomes even closer, it should be outside; or it should be inside
	//you have all the insided tris, then ...
	//divergence theorem to locate the volume

	int num_point_inside_1 = 0, num_point_inside_2 = 0;  
	int num_relevant_face_1 = 0, num_relevant_face_2 = 0;
	for (i = 0;i<nV_2;++i) {   //index of points that need to know if inside mesh
		num_ray_intersect = 0;
		for (j=0;j<nV_1;++j) {  //index of triangles
			if (((vertices_y_2[i]>face_coord_1[9*j+1])&&(vertices_y_2[i]>face_coord_1[9*j+4])&&(vertices_y_2[i]>face_coord_1[9*j+7])) ||
			   ((vertices_y_2[i]<face_coord_1[9*j+1])&&(vertices_y_2[i]<face_coord_1[9*j+4])&&(vertices_y_2[i]<face_coord_1[9*j+7])) ||
			   ((vertices_z_2[i]>face_coord_1[9*j+2])&&(vertices_y_2[i]>face_coord_1[9*j+5])&&(vertices_y_2[i]>face_coord_1[9*j+8])) ||
                           ((vertices_z_2[i]<face_coord_1[9*j+2])&&(vertices_y_2[i]<face_coord_1[9*j+5])&&(vertices_y_2[i]<face_coord_1[9*j+8]))) {
				continue;
			}
			else if (intersect(i, j, vertices_x_2, vertices_y_2, vertices_z_2, face_coord_1)) {
				num_ray_intersect++;
			}
		}
		if (num_ray_intersect%2==1) { inside_point_set_2[num_point_inside_2] = i; num_point_inside_2++; }
	}

	for (i = 0;i<nV_1;++i) {   //index of points that need to know if inside mesh
                num_ray_intersect = 0;
                for (j=0;j<nV_2;++j) {  //index of triangles
                        if (((vertices_y_1[i]>face_coord_2[9*j+1])&&(vertices_y_1[i]>face_coord_2[9*j+4])&&(vertices_y_1[i]>face_coord_2[9*j+7])) ||
                           ((vertices_y_1[i]<face_coord_2[9*j+1])&&(vertices_y_1[i]<face_coord_2[9*j+4])&&(vertices_y_1[i]<face_coord_2[9*j+7])) ||
                           ((vertices_z_1[i]>face_coord_2[9*j+2])&&(vertices_y_1[i]>face_coord_2[9*j+5])&&(vertices_y_1[i]>face_coord_2[9*j+8])) ||
                           ((vertices_z_1[i]<face_coord_2[9*j+2])&&(vertices_y_1[i]<face_coord_2[9*j+5])&&(vertices_y_1[i]<face_coord_2[9*j+8]))) {
                                continue;
                        }
                        else if (intersect(i, j, vertices_x_1, vertices_y_1, vertices_z_1, face_coord_2)) {
                                num_ray_intersect++;
                        }
                }
                if (num_ray_intersect%2==1) { inside_point_set_1[num_point_inside_1] = i; num_point_inside_1++; }
        }

	double *relevant_face_1 = (double *)malloc(sizeof(double)*nF_1*9);
        double *relevant_face_2 = (double *)malloc(sizeof(double)*nF_2*9);

	for (i=0;i<nF_1;++i) {
		for (j=0;j<num_point_inside_1;++j) {	
			if ((inside_point_set_1[j]==faces_1[3*i])||(inside_point_set_1[j]==faces_1[3*i+1])||(inside_point_set_1[j]==faces_1[3*i+2])) {
				memcpy(&(relevant_face_1[9*num_relevant_face_1]), &(face_coord_1[9*i]), sizeof(double)*9);
				num_relevant_face_1++;
				break;
			}
		}
	}

	
	for (i=0;i<nF_2;++i) {
                for (j=0;j<num_point_inside_2;++j) {
                        if ((inside_point_set_2[j]==faces_2[3*i])||(inside_point_set_2[j]==faces_2[3*i+1])||(inside_point_set_2[j]==faces_2[3*i+2])) {
                                memcpy(&(relevant_face_2[9*num_relevant_face_2]), &(face_coord_2[9*i]), sizeof(double)*9);
                                num_relevant_face_2++;
                                break;
                        }
                }
        }

	double vol = get_volume(num_relevant_face_1, num_relevant_face_2, relevant_face_1, relevant_face_2);

  	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
  	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);   
 

	printf("%d\n%d\n", nV_1, nF_1);


	//free resources 
//	free(in); free(out); free(cuda_out);
	return 0;
}
