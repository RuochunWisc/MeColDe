// Reference Scan implementation - Author: Ananoymous student of ME759 Fall 2017
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/scan.h>


void initializeSize(FILE* fp, int* nVertices, int* nFaces)
{
	char x[1024];
	while (fscanf(fp, "%1023s", x)) {
        	if (strcmp(x,"ply")==0) continue;
		if ((strcmp(x,"format")==0)||(strcmp(x,"comment")==0)||(strcmp(x,"property")==0)) {fgets(x, 1023, fp);continue;}
								//	{fscanf(fp,"%[^\n]", x);continue;}
		if (strcmp(x,"element")==0) {
			fscanf(fp, "%1023s", x);
			if (strcmp(x,"vertex")==0) fscanf(fp, "%d\n", &nVertices[0]);
			else if (strcmp(x,"face")==0) fscanf(fp, "%d\n", &nFaces[0]);
		}
		if (strcmp(x,"end_header")==0) break;
    	}
}

void initializeArr(FILE* fp, float* ArrVerX, float ArrVerY, float* ArrVerZ, float* ArrFace, int V, int F)
{
	int i;
	char x[1024];	
	for(i=0; i<V; ++i){
		fscanf(fp,"%f",&ArrVerX[i]);
		fscanf(fp,"%f",&ArrVerY[i]);
		fscanf(fp,"%f",&ArrVerZ[i]);
		fgets(x, 1023, fp);
//		if(r == EOF){
//			rewind(fp);
//		}
	}

	for (i=0; i<F; ++i){
		fscanf(fp, "%1023s", x);
		fscanf(fp, "%f", &ArrFace[3*i]);
		fscanf(fp, "%f", &ArrFace[3*i+1]);
		fscanf(fp, "%f", &ArrFace[3*i+2]);
	}
}



int main(int argc, char* argv[]) {
	FILE *fp = fopen("bun_zipper.ply","r");
	//allocate resources
	int nV=0, nF=0;
	float time = 0.f;
	initializeSize(fp,&nV, &nF);
	float *vertices_x= (float *)malloc(sizeof(float)*nV);
	float *vertices_y= (float *)malloc(sizeof(float)*nV);
	float *vertices_z= (float *)malloc(sizeof(float)*nV);
        float *faces   = (float *)malloc(sizeof(float)*nF*3);
	initializeArr(fp,vertices_x,vertices_y,vertices_z,faces,nV,nF);
	// Your code here
 
  
 // hipMalloc((void**)&dout,size);
	//hipMalloc((void**)&din,size);
 
  hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
  hipEventRecord(startEvent_inc,0); // starting timing for inclusive  
	//AABB construction
	//divergence theorem to locate the volume  

	
  hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
  hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);   
 

	printf("%d\n%d\n", nV, nF);


	//free resources 
//	free(in); free(out); free(cuda_out);
	return 0;
}
