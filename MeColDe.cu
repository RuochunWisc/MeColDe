#include "hip/hip_runtime.h"
// Reference Scan implementation - Author: Ananoymous student of ME759 Fall 2017
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/scan.h>


void initializeSize(FILE* fp, int* nVertices, int* nFaces)
{
	char x[1024];
	while (fscanf(fp, "%1023s", x)) {
        	if (strcmp(x,"ply")==0) continue;
		if ((strcmp(x,"format")==0)||(strcmp(x,"comment")==0)||(strcmp(x,"property")==0)) {fgets(x, 1023, fp);continue;}
								//	{fscanf(fp,"%[^\n]", x);continue;}
		if (strcmp(x,"element")==0) {
			fscanf(fp, "%1023s", x);
			if (strcmp(x,"vertex")==0) fscanf(fp, "%d\n", &nVertices[0]);
			else if (strcmp(x,"face")==0) fscanf(fp, "%d\n", &nFaces[0]);
			else {fgets(x, 1023, fp);continue;}
		}
		if (strcmp(x,"end_header")==0) break;
    	}
}

void initializeArr(FILE* fp, double* ArrVerX, double* ArrVerY, double* ArrVerZ, int* ArrFace, int V, int F)
{
	int i;
	char x[1024];	
	for(i=0; i<V; ++i){
		fscanf(fp,"%lf",&ArrVerX[i]);
		fscanf(fp,"%lf",&ArrVerY[i]);
		fscanf(fp,"%lf",&ArrVerZ[i]);
		fgets(x, 1023, fp);
//		if(r == EOF){
//			rewind(fp);
//		}
	}

	for (i=0; i<F; ++i){
		fscanf(fp, "%1023s", x);
		fscanf(fp, "%d", &ArrFace[3*i]);
		fscanf(fp, "%d", &ArrFace[3*i+1]);
		fscanf(fp, "%d", &ArrFace[3*i+2]);
	}
}

void cross(double* res, double* u, double* v) 
{
	res[0] = u[1]*v[2] - u[2]*v[1];
	res[1] = u[2]*v[0] - u[0]*v[2];
	res[2] = u[0]*v[1] - u[1]*v[0];
}

/*
__device__ void d_cross(double* res, double* u, double* v)
{
        res[0] = u[1]*v[2] - u[2]*v[1];
        res[1] = u[2]*v[0] - u[0]*v[2];
        res[2] = u[0]*v[1] - u[1]*v[0];
}
*/



double dot(double* u, double* v)
{
	return u[0]*v[0]+u[1]*v[1]+u[2]*v[2];
}

bool intersect(double *p_x, double *p_y, double *p_z, double *tri)
{
	double small_num = 0.00000001;
	double e1[3] = {tri[3]-tri[0], tri[4]-tri[1], tri[5]-tri[2]};
	double e2[3] = {tri[6]-tri[0], tri[7]-tri[1], tri[8]-tri[2]};

//	double norm = sqrt(e1[0]*e1[0]+e1[1]*e1[1]+e1[2]*e1[2]);
//	if (norm == 0.0) {printf("oooh");}

	double d[3] = {0,1,0};
	double h[3];
	cross(h, d, e2);
	double a = dot(e1, h);
	if ((a>(-1*small_num))&&(a<small_num)) return false;
	double f = 1/a;
	double s[3] = {*p_x-tri[0], *p_y-tri[1], *p_z-tri[2]};
	double u = f*(dot(s, h));
	if ((u<0.0)||(u>1.0)) return false;
	double q[3];
	cross(q, s, e1);
	double v = f*(dot(d, q));
	if ((v<0.0)||(u+v>1.0)) return false;
	double t = f*(dot(e2, q));
	if (t>small_num) return true;
	else return false;
}

__global__ void find_volume_normal(int tot_num, double *faces, double *volume, double *normal) {
    	int yourID = blockIdx.x*blockDim.x+threadIdx.x;
	double *yourTri;
	yourTri = faces + 9*yourID;
	if (yourID<tot_num) {
		double d13[3] = {yourTri[3]-yourTri[6], yourTri[4]-yourTri[7], yourTri[5]-yourTri[8]};
		double d12[3] = {yourTri[0]-yourTri[3], yourTri[1]-yourTri[4], yourTri[2]-yourTri[5]};
		double cr[3];
		//d_cross(cr, d13, d12);
		cr[0] = d13[1]*d12[2] - d13[2]*d12[1];
        	cr[1] = d13[2]*d12[0] - d13[0]*d12[2];
        	cr[2] = d13[0]*d12[1] - d13[1]*d12[0];
		
		double crNorm = sqrt(cr[0]*cr[0]+cr[1]*cr[1]+cr[2]*cr[2]);

//		if (crNorm==0.0) crNorm = 1;	

		double area = 0.5*crNorm;
		double zMean = (yourTri[2]+yourTri[5]+yourTri[8])/3;
		double nz = (-1)*cr[2]/crNorm;
		volume[yourID] = area*zMean*nz;
		normal[3*yourID] = cr[0]/crNorm;
		normal[3*yourID+1] = cr[1]/crNorm;
		normal[3*yourID+2] = cr[2]/crNorm;
	}
}

double get_volume(int num_1, int num_2, double* faces_1, double* faces_2, double* normal_1)
{

	double *d_faces;  
	hipMalloc(&d_faces, sizeof(double)*(num_1+num_2)*9);

	double *d_vol, *d_nor;
	hipMalloc(&d_vol, sizeof(double)*(num_1+num_2)*1);
	hipMalloc(&d_nor, sizeof(double)*(num_1+num_2)*3);

	double *h_vol = (double *)malloc(sizeof(double)*(num_1+num_2)*1);
	double *h_nor = (double *)malloc(sizeof(double)*(num_1+num_2)*3);

	hipMemcpy(d_faces,faces_1,sizeof(double)*num_1*9,hipMemcpyHostToDevice);
	hipMemcpy(d_faces+num_1*9,faces_2,sizeof(double)*num_2*9,hipMemcpyHostToDevice);
  	find_volume_normal<<<(num_1+num_2+1023)/1024, 1024>>>(num_1+num_2, d_faces, d_vol, d_nor);
  	hipMemcpy(h_vol, d_vol, sizeof(double)*(num_1+num_2), hipMemcpyDeviceToHost);
	hipMemcpy(h_nor, d_nor, sizeof(double)*(num_1+num_2)*3,  hipMemcpyDeviceToHost);

	double sum_v = 0.0, nor_x_tot = 0.0, nor_y_tot = 0.0, nor_z_tot = 0.0;
	int i;
	for (i = 0; i<num_1+num_2; ++i) {
		sum_v = sum_v + h_vol[i];
	}

	for (i = 0; i<num_1; ++i) {
		nor_x_tot += h_nor[3*i];
		nor_y_tot += h_nor[3*i+1];
		nor_z_tot += h_nor[3*i+2];
	}

	normal_1[0] = nor_x_tot/num_1;
	normal_1[1] = nor_y_tot/num_1;
	normal_1[2] = nor_z_tot/num_1;
	return sum_v;
}

int main(int argc, char* argv[]) {
	FILE *fp_1 = fopen("bun_zipper.ply","r");
	FILE *fp_2 = fopen("test_mesh.ply","r");
	//allocate resources
	int nV_1=0, nF_1=0, nV_2=0, nF_2=0,i,j;
	int num_ray_intersect;
	float time = 0.f;
	initializeSize(fp_1,&nV_1, &nF_1);
	initializeSize(fp_2,&nV_2, &nF_2);
	double *vertices_x_1= (double *)malloc(sizeof(double)*nV_1);
	double *vertices_y_1= (double *)malloc(sizeof(double)*nV_1);
	double *vertices_z_1= (double *)malloc(sizeof(double)*nV_1);
        int *faces_1   = (int *)malloc(sizeof(int)*nF_1*3);
	double *vertices_x_2= (double *)malloc(sizeof(double)*nV_2);
        double *vertices_y_2= (double *)malloc(sizeof(double)*nV_2);
        double *vertices_z_2= (double *)malloc(sizeof(double)*nV_2);
        int *faces_2   = (int *)malloc(sizeof(int)*nF_2*3);

	initializeArr(fp_1,vertices_x_1,vertices_y_1,vertices_z_1,faces_1,nV_1,nF_1);
	initializeArr(fp_2,vertices_x_2,vertices_y_2,vertices_z_2,faces_2,nV_2,nF_2);

// for test reasons
//	double dis = 0.05;
//	for (i=0;i<nV_2;++i) {
//		vertices_x_2[i] += dis;
//		vertices_y_2[i] += dis;
//		vertices_z_2[i] += dis;
//	}

//	printf("%f\n%d\n", *vertices_x_1, *faces_2);

//end of test

 
  	double *face_coord_1 = (double *)malloc(sizeof(double)*nF_1*9);
	double *face_coord_2 = (double *)malloc(sizeof(double)*nF_2*9);
 // hipMalloc((void**)&dout,size);
	//hipMalloc((void**)&din,size);
	int *inside_point_set_1 = (int *)malloc(sizeof(int)*nV_1);
	int *inside_point_set_2 = (int *)malloc(sizeof(int)*nV_2);
 
  	hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
  	hipEventRecord(startEvent_inc,0); // starting timing for inclusive  

	for (i=0;i<nF_1;++i){
		face_coord_1[9*i] = vertices_x_1[faces_1[3*i]];
		face_coord_1[9*i+1] = vertices_y_1[faces_1[3*i]];
  		face_coord_1[9*i+2] = vertices_z_1[faces_1[3*i]];
		face_coord_1[9*i+3] = vertices_x_1[faces_1[3*i+1]];
                face_coord_1[9*i+4] = vertices_y_1[faces_1[3*i+1]];
                face_coord_1[9*i+5] = vertices_z_1[faces_1[3*i+1]];
		face_coord_1[9*i+6] = vertices_x_1[faces_1[3*i+2]];
                face_coord_1[9*i+7] = vertices_y_1[faces_1[3*i+2]];
                face_coord_1[9*i+8] = vertices_z_1[faces_1[3*i+2]];
	}

	for (i=0;i<nF_2;++i){
                face_coord_2[9*i] = vertices_x_2[faces_2[3*i]];
                face_coord_2[9*i+1] = vertices_y_2[faces_2[3*i]];
                face_coord_2[9*i+2] = vertices_z_2[faces_2[3*i]];
                face_coord_2[9*i+3] = vertices_x_2[faces_2[3*i+1]];
                face_coord_2[9*i+4] = vertices_y_2[faces_2[3*i+1]];
                face_coord_2[9*i+5] = vertices_z_2[faces_2[3*i+1]];
                face_coord_2[9*i+6] = vertices_x_2[faces_2[3*i+2]];
                face_coord_2[9*i+7] = vertices_y_2[faces_2[3*i+2]];
                face_coord_2[9*i+8] = vertices_z_2[faces_2[3*i+2]];
        }
	//AABB construction
	//NO, dont do this; instead, find points if inside the mesh to determine if this point belongs to the intersection part, this is not likely to be wrong in such a convex struct
	//how to determine if a point is inside the mesh? a lite way, find the closet point on the mesh(para), then move the point towards the direction defined by the point the center of the mesh(the dist to move is around 1/10 of the element length), if the closet point becomes even closer, it should be outside; or it should be inside
	//you have all the insided tris, then ...
	//divergence theorem to locate the volume

	int num_point_inside_1 = 0, num_point_inside_2 = 0;  
	int num_relevant_face_1 = 0, num_relevant_face_2 = 0;
	for (i = 0;i<nV_2;++i) {   //index of points that need to know if inside mesh
		num_ray_intersect = 0;
		for (j=0;j<nF_1;++j) {  //index of triangles
			if (((vertices_x_2[i]>face_coord_1[9*j+0])&&(vertices_x_2[i]>face_coord_1[9*j+3])&&(vertices_x_2[i]>face_coord_1[9*j+6])) ||
			   ((vertices_x_2[i]<face_coord_1[9*j+0])&&(vertices_x_2[i]<face_coord_1[9*j+3])&&(vertices_x_2[i]<face_coord_1[9*j+6])) ||
			   ((vertices_z_2[i]>face_coord_1[9*j+2])&&(vertices_z_2[i]>face_coord_1[9*j+5])&&(vertices_z_2[i]>face_coord_1[9*j+8])) ||
                           ((vertices_z_2[i]<face_coord_1[9*j+2])&&(vertices_z_2[i]<face_coord_1[9*j+5])&&(vertices_z_2[i]<face_coord_1[9*j+8]))) {
				continue;
			}
			else if (intersect(vertices_x_2+i, vertices_y_2+i, vertices_z_2+i, face_coord_1+9*j)) {
				num_ray_intersect++;
			}
		}
		if (num_ray_intersect%2==1) { inside_point_set_2[num_point_inside_2] = i; num_point_inside_2++; }
	}

	for (i = 0;i<nV_1;++i) {   //index of points that need to know if inside mesh
                num_ray_intersect = 0;
                for (j=0;j<nF_2;++j) {  //index of triangles
                        if (((vertices_x_1[i]>face_coord_2[9*j+0])&&(vertices_x_1[i]>face_coord_2[9*j+3])&&(vertices_x_1[i]>face_coord_2[9*j+6])) ||
                           ((vertices_x_1[i]<face_coord_2[9*j+0])&&(vertices_x_1[i]<face_coord_2[9*j+3])&&(vertices_x_1[i]<face_coord_2[9*j+6])) ||
                           ((vertices_z_1[i]>face_coord_2[9*j+2])&&(vertices_z_1[i]>face_coord_2[9*j+5])&&(vertices_z_1[i]>face_coord_2[9*j+8])) ||
                           ((vertices_z_1[i]<face_coord_2[9*j+2])&&(vertices_z_1[i]<face_coord_2[9*j+5])&&(vertices_z_1[i]<face_coord_2[9*j+8]))) {
                                continue;
                        }
                        else if (intersect(vertices_x_1+i, vertices_y_1+i, vertices_z_1+i, face_coord_2+9*j)) {
                                num_ray_intersect++;
                        }
                }
                if (num_ray_intersect%2==1) { inside_point_set_1[num_point_inside_1] = i; num_point_inside_1++; }
        }


	double x_tot = 0.0, y_tot = 0.0, z_tot = 0.0;
	for (i=0;i<num_point_inside_1;++i) {
		x_tot += vertices_x_1[inside_point_set_1[i]];
		y_tot += vertices_y_1[inside_point_set_1[i]];
		z_tot += vertices_y_1[inside_point_set_1[i]];
	}

	for (i=0;i<num_point_inside_2;++i) {
                x_tot += vertices_x_2[inside_point_set_2[i]];
                y_tot += vertices_y_2[inside_point_set_2[i]];
                z_tot += vertices_z_2[inside_point_set_2[i]];
        }

	double center[3] = {x_tot/(num_point_inside_1+num_point_inside_2), 
				y_tot/(num_point_inside_1+num_point_inside_2), 
				z_tot/(num_point_inside_1+num_point_inside_2)};

	double *relevant_face_1 = (double *)malloc(sizeof(double)*nF_1*9);
        double *relevant_face_2 = (double *)malloc(sizeof(double)*nF_2*9);

	for (i=0;i<nF_1;++i) {
		for (j=0;j<num_point_inside_1;++j) {	
			if ((inside_point_set_1[j]==faces_1[3*i])||(inside_point_set_1[j]==faces_1[3*i+1])||(inside_point_set_1[j]==faces_1[3*i+2])) {
				memcpy(relevant_face_1+9*num_relevant_face_1, face_coord_1+9*i, sizeof(double)*9);
				num_relevant_face_1++;
				break;
			}
		}
	}

	
	for (i=0;i<nF_2;++i) {
                for (j=0;j<num_point_inside_2;++j) {
                        if ((inside_point_set_2[j]==faces_2[3*i])||(inside_point_set_2[j]==faces_2[3*i+1])||(inside_point_set_2[j]==faces_2[3*i+2])) {
                                memcpy(relevant_face_2+9*num_relevant_face_2, face_coord_2+9*i, sizeof(double)*9);
                                num_relevant_face_2++;
                                break;
                        }
                }
        }
	
	printf("%d   %d\n",num_point_inside_1, num_point_inside_2);
	printf("%d   %d\n",num_relevant_face_1, num_relevant_face_2);
	printf("%f\n", *relevant_face_1); 	


	double *normal_1 = (double *)malloc(sizeof(double)*3);

	double vol = get_volume(num_relevant_face_1, num_relevant_face_2, relevant_face_1, relevant_face_2, normal_1);

  	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
  	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);   
 

	printf("%d\n%d\n", nV_1, nF_1);
	printf("center is %lf %lf %lf\n", center[0], center[1], center[2]);
	printf("normal of one direction is %lf %lf %lf\n", normal_1[0], normal_1[1], normal_1[2]);
	printf("%lf\n", vol);
	//free resources 
//	free(in); free(out); free(cuda_out);
	return 0;
}
