// Reference Scan implementation - Author: Ananoymous student of ME759 Fall 2017
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/scan.h>


void initializeArray(FILE* fp, float* ArrVer, float* ArrFace, int nVertices, int nFaces)
{
	char x[1024];
	long int tot_line = 0;
	while (fscanf(fp, "%1023s", x)) {
		tot_line++;
		if (tot_line > 20) break;
		puts(x);
        	if (x=="ply") continue;
		if ((x=="format")||(x=="comment")||(x=="property")) {fgets(x, 1023, fp);continue;}
								//	{fscanf(fp,"%[^\n]", x);continue;}
		if (x=="element") {
			fscanf(fp, "%1023s", x);
			if (x=="vertex") fscanf(fp, "%d", nVertices);
			else if (x=="face") fscanf(fp, "%d", nFaces);
		}
    	}
	
//	for( int i=0; i<nElements; i++){
//		int r=fscanf(fp,"%f",&arr[i]);
//		if(r == EOF){
//			rewind(fp);
//		}
//	}
}



int main(int argc, char* argv[]) {
	FILE *fp = fopen("bun_zipper.ply","r");
	//allocate resources
	int nV=0,nF=0;
	float* vertices, *faces;
	float time = 0.f;
	initializeArray(fp,vertices, faces,nV, nF);
	// Your code here
 
  
 // hipMalloc((void**)&dout,size);
	//hipMalloc((void**)&din,size);
 
  hipEvent_t startEvent_inc, stopEvent_inc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
  hipEventRecord(startEvent_inc,0); // starting timing for inclusive  
  

	
  hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
  hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&time, startEvent_inc, stopEvent_inc);   
 

	printf("%d\n%d\n",nV,nF);


	//free resources 
//	free(in); free(out); free(cuda_out);
	return 0;
}
